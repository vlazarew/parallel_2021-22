#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
// #include <mpi/mpi.h>
#include <mpi.h>
#include <string>
#include <cstring>
#include <cmath>
#include <vector>

using namespace std;

#pragma region Entities and constants
const int MAIN_PROCESS_ID = 0;
const bool USE_GPU = true;
// Размер GPU блока
const int GPU_BLOCK_SIZE = 512;

// Оси разбиения
enum Axis
{
    X,
    Y,
    Z,
};

// Исходный параллелпипед
struct Parallelepiped
{
    // Размеры параллелепипеда по каждой из осей
    double x;
    double y;
    double z;
};

// Параллелпипед, обрабтываемый отдельным процессом
struct ProcessParallelepiped
{
    // Границы по оси X
    int xMin, xMax;

    // Границы по оси Y
    int yMin, yMax;

    // Границы по оси Z
    int zMin, zMax;

    // Размеры по каждой из осей + объем области
    int dx, dy, dz, size;
};

// Шаги пространственной сетки по каждой из осей
struct GridSteps
{
    double x;
    double y;
    double z;
};

struct SolverVariables
{
    // Исходный параллелпипед
    Parallelepiped L;
    // T - итоговое время
    double T;
    // N - количество точек пространственной сетки / K - количество точек временной сетки / steps - кол-во шагов для решения
    int N, K, steps;

    // Шаги пространственной сетки по каждой из осей
    GridSteps H;
    // Шаг временной сетки
    double tau;
    // Размер слоя
    int layerSize;

    // Id процесса / количество MPI-процессов
    int processId, countOfProcesses;

    // Параллелпипед, обрабтываемый отдельным процессом
    ProcessParallelepiped processParallelepiped;
    // Параллелепипеды-соседи на передачу
    vector<ProcessParallelepiped> send;
    // Параллелепипеды-соседи на прием
    vector<ProcessParallelepiped> recv;
    // Параллелепипеды-соседи в формате id процесса
    vector<int> processIds;
};
#pragma endregion

#pragma region Handle and init arguments

float getFloatValueFromArg(const char *key, int argc, char *argv[], float defaultValue)
{
    float value = 0;
    bool valueFound = false;
    // Первый параметр - ссылка на сборку
    for (int i = 1; i < argc; i++)
    {
        string currentArgument(argv[i]);
        int argName = currentArgument.find(key);
        if (argName != string::npos)
        {
            string argString = currentArgument.substr(argName + strlen(key));
            try
            {
                value = atof(argString.c_str());
                valueFound = true;
                break;
            }
            catch (...)
            {
                // Не поддерживается на кластерах
                // throw runtime_error("Invalid input epsilon");
                throw;
            }
        }
    }

    return valueFound ? value : defaultValue;
}

int getIntValueFromArg(const char *key, int argc, char *argv[], int defaultValue)
{
    int value = 0;
    bool valueFound = false;
    // Первый параметр - ссылка на сборку
    for (int i = 1; i < argc; i++)
    {
        string currentArgument(argv[i]);
        int argName = currentArgument.find(key);
        if (argName != string::npos)
        {
            string argString = currentArgument.substr(argName + strlen(key));
            try
            {
                value = atoi(argString.c_str());
                valueFound = true;
                break;
            }
            catch (...)
            {
                // Не поддерживается на кластерах
                // throw runtime_error("Invalid input epsilon");
                throw;
            }
        }
    }

    return valueFound ? value : defaultValue;
}

void initVariables(int argc, char *argv[], SolverVariables &variables, int processId, int countOfProcesses)
{
    // Первый параметр - ссылка на сборку
    variables.L.x = getFloatValueFromArg("-Lx=", argc, argv, 1);
    variables.L.y = getFloatValueFromArg("-Ly=", argc, argv, 1);
    variables.L.z = getFloatValueFromArg("-Lz=", argc, argv, 1);
    variables.T = getFloatValueFromArg("-T=", argc, argv, 1);
    variables.N = getIntValueFromArg("-N=", argc, argv, 128);
    variables.K = getIntValueFromArg("-K=", argc, argv, 2000);
    variables.steps = getIntValueFromArg("-steps=", argc, argv, 5);
    // Остальные параметры будут игнорироваться (ну или позже добавлю какие-нибудь свои кастомные)

    variables.H.x = variables.L.x / variables.N;
    variables.H.y = variables.L.y / variables.N;
    variables.H.z = variables.L.z / variables.N;
    variables.tau = variables.T / variables.K;
    variables.layerSize = pow(variables.N + 1, 3);

    variables.processId = processId;
    variables.countOfProcesses = countOfProcesses;
}

#pragma endregion

#pragma region Split parallelepiped

// Конструктор параллелепипеда по макс/мин координатам
ProcessParallelepiped createParallelepiped(int xMin, int xMax, int yMin, int yMax, int zMin, int zMax)
{
    int dx = xMax - xMin + 1;
    int dy = yMax - yMin + 1;
    int dz = zMax - zMin + 1;
    int size = dx * dy * dz;

    return ProcessParallelepiped{xMin, xMax, yMin, yMax, zMin, zMax, dx, dy, dz, size};
}

// Блочное разбиение параллелепипеда
void makeSplit(int xMin, int xMax, int yMin, int yMax, int zMin, int zMax, int countOfProcesses, Axis axis,
               vector<ProcessParallelepiped> &subParallelepipeds)
{

    // Если один процесс, то ему достается полный параллелепипед
    if (countOfProcesses == 1)
    {
        subParallelepipeds.push_back(createParallelepiped(xMin, xMax, yMin, yMax, zMin, zMax));
        return;
    }

    // Делаем область по текущей оси, делаем паралл, разбиваем дальше
    if (countOfProcesses % 2 == 1)
    {
        if (axis == X)
        {
            int newXMax = xMin + (xMax - xMin) / countOfProcesses;
            subParallelepipeds.push_back(createParallelepiped(xMin, newXMax, yMin, yMax, zMin, zMax));

            xMin = newXMax + 1;
            axis = Y;
        }
        else if (axis == Y)
        {
            int newYMax = yMin + (yMax - yMin) / countOfProcesses;
            subParallelepipeds.push_back(createParallelepiped(xMin, xMax, yMin, newYMax, zMin, zMax));

            yMin = newYMax + 1;
            axis = Z;
        }
        else if (axis == Z)
        {
            int newZMax = zMin + (zMax - zMin) / countOfProcesses;
            subParallelepipeds.push_back(createParallelepiped(xMin, xMax, yMin, yMax, zMin, newZMax));

            zMin = newZMax + 1;
            axis = X;
        }

        countOfProcesses--;
    }

    // Для оси делим область пополам и запускаем рекурсию для sub-параллелепипедов
    int newCountOfProcesses = countOfProcesses / 2;
    if (axis == X)
    {
        int newXMax = (xMin + xMax) / 2;
        makeSplit(xMin, newXMax, yMin, yMax, zMin, zMax, newCountOfProcesses, Y, subParallelepipeds);
        makeSplit(newXMax + 1, xMax, yMin, yMax, zMin, zMax, newCountOfProcesses, Y, subParallelepipeds);
    }
    else if (axis == Y)
    {
        int newYMax = (yMin + yMax) / 2;
        makeSplit(xMin, xMax, yMin, newYMax, zMin, zMax, newCountOfProcesses, Z, subParallelepipeds);
        makeSplit(xMin, xMax, newYMax + 1, yMax, zMin, zMax, newCountOfProcesses, Z, subParallelepipeds);
    }
    else if (axis == Z)
    {
        int newZMax = (zMin + zMax) / 2;
        makeSplit(xMin, xMax, yMin, yMax, zMin, newZMax, newCountOfProcesses, X, subParallelepipeds);
        makeSplit(xMin, xMax, yMin, yMax, newZMax + 1, zMax, newCountOfProcesses, X, subParallelepipeds);
    }
}

// Запуск разбиения параллелпипеда на более мелкие
vector<ProcessParallelepiped> splitParallelepiped(int N, int countOfProcesses)
{
    vector<ProcessParallelepiped> subParallelepipeds;

    // Коэффициенты - параметры исходного параллелепипеда
    makeSplit(0, N, 0, N, 0, N, countOfProcesses, X, subParallelepipeds);

    return subParallelepipeds;
}

#pragma endregion

#pragma region Actions with neighbours

// Проверка на то, что первый параллелепипед (координаты с 1) находится внутри второго (координаты с 2)
bool isInside(int xMin1, int xMax1, int yMin1, int yMax1, int xMin2, int xMax2, int yMin2, int yMax2)
{
    return xMin2 <= xMin1 && xMax1 <= xMax2 && yMin2 <= yMin1 && yMax1 <= yMax2;
}

// Получить соприкосновение двух параллелипедов (true - если соприкосновение есть, иначе - false). result - соприкосновение
bool getNeighbours(const ProcessParallelepiped &first, const ProcessParallelepiped &second, ProcessParallelepiped &result)
{
    // Если начало одного == конец второго по оси X
    if (first.xMin == second.xMax + 1 || second.xMin == first.xMax + 1)
    {
        // Берем точку соприкосновения двух параллелепипедов
        int x = (first.xMin == second.xMax + 1) ? first.xMin : first.xMax;

        // Если fisrt внутри second
        if (isInside(first.yMin, first.yMax, first.zMin, first.zMax, second.yMin, second.yMax, second.zMin,
                     second.zMax))
        {
            // Создадим параллепипед по X размером 1 (соприкосновение блоков) и размерами first => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(x, x, first.yMin, first.yMax, first.zMin, first.zMax);
            return true;
        }

        // Если second внутри first
        if (isInside(second.yMin, second.yMax, second.zMin, second.zMax, first.yMin, first.yMax, first.zMin,
                     first.zMax))
        {
            // Создадим параллепипед по X размером 1 (соприкосновение блоков) и размерами second => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(x, x, second.yMin, second.yMax, second.zMin, second.zMax);
            return true;
        }

        return false;
    }

    // Если начало одного == конец второго по оси Y
    if (first.yMin == second.yMax + 1 || second.yMin == first.yMax + 1)
    {
        // Берем точку соприкосновения двух параллелепипедов
        int y = (first.yMin == second.yMax + 1) ? first.yMin : first.yMax;

        // Если fisrt внутри second
        if (isInside(first.xMin, first.xMax, first.zMin, first.zMax, second.xMin, second.xMax, second.zMin,
                     second.zMax))
        {
            // Создадим параллепипед по Y размером 1 (соприкосновение блоков) и размерами first => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(first.xMin, first.xMax, y, y, first.zMin, first.zMax);
            return true;
        }

        // Если second внутри first
        if (isInside(second.xMin, second.xMax, second.zMin, second.zMax, first.xMin, first.xMax, first.zMin,
                     first.zMax))
        {
            // Создадим параллепипед по Y размером 1 (соприкосновение блоков) и размерами second => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(second.xMin, second.xMax, y, y, second.zMin, second.zMax);
            return true;
        }

        return false;
    }

    // Если начало одного == конец второго по оси Z
    if (first.zMin == second.zMax + 1 || second.zMin == first.zMax + 1)
    {
        // Берем точку соприкосновения двух параллелепипедов
        int z = (first.zMin == second.zMax + 1) ? first.zMin : first.zMax;

        // Если fisrt внутри second
        if (isInside(first.xMin, first.xMax, first.yMin, first.yMax, second.xMin, second.xMax, second.yMin,
                     second.yMax))
        {
            // Создадим параллепипед по Z размером 1 (соприкосновение блоков) и размерами first => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(first.xMin, first.xMax, first.yMin, first.yMax, z, z);
            return true;
        }

        // Если second внутри first
        if (isInside(second.xMin, second.xMax, second.yMin, second.yMax, first.xMin, first.xMax, first.yMin,
                     first.yMax))
        {
            // Создадим параллепипед по Z размером 1 (соприкосновение блоков) и размерами second => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(second.xMin, second.xMax, second.yMin, second.yMax, z, z);
            return true;
        }

        return false;
    }

    return false;
}

// Заполняем соседей
void fillNeighbours(vector<ProcessParallelepiped> &parallelepipeds, SolverVariables &variables)
{
    variables.send.clear();
    variables.recv.clear();
    variables.processIds.clear();
    ProcessParallelepiped targetParallelepiped = variables.processParallelepiped;

    for (int i = 0; i < variables.countOfProcesses; i++)
    {
        // Самого себя не обрабатываем
        if (i == variables.processId)
        {
            continue;
        }

        ProcessParallelepiped send;
        ProcessParallelepiped recv;

        ProcessParallelepiped processParallelepiped = parallelepipeds[i];
        // Получаем соседей, которым будем отправлять информацию
        if (!getNeighbours(targetParallelepiped, processParallelepiped, send))
        {
            continue;
        }

        // Получаем соседей, от которых будем получать информацию
        getNeighbours(processParallelepiped, targetParallelepiped, recv);
        variables.processIds.push_back(i);
        variables.send.push_back(send);
        variables.recv.push_back(recv);
    }
}

#pragma endregion

#pragma region Math calculations

// Аналитическое решение
__host__ __device__ double getAnalyticValue(double x, double y, double z, double t, const Parallelepiped &L)
{
    double at = M_PI * sqrt(1 / pow(L.x, 2) + 1 / pow(L.y, 2) + 4 / pow(L.z, 2));

    return sin(M_PI * x / L.x) * sin(M_PI * y / L.y) * sin(2 * z * M_PI / L.z) * cos(at * t + 2 * M_PI);
}

// Начальные условия
__host__ __device__ double getPhi(double x, double y, double z, const Parallelepiped &L)
{
    return getAnalyticValue(x, y, z, 0, L);
}

// Получить индекс по x, y, z для конкрентного параллелепипеда
__host__ __device__ int getIndex(int x, int y, int z, const ProcessParallelepiped &target)
{
    return (x - target.xMin) * target.dy * target.dz + (y - target.yMin) * target.dz + (z - target.zMin);
}

// Получить локальный индекс
__host__ __device__ int getLocalIndex(int x, int y, int z, const SolverVariables &variables)
{
    return getIndex(x, y, z, variables.processParallelepiped);
}

__host__ __device__ double findValue(int size, const double *u, int x, int y, int z, const double *recv, const ProcessParallelepiped *recvNeighboursData, const SolverVariables &variables)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    if (processParallelepiped.xMin <= x && processParallelepiped.xMax >= x &&
        processParallelepiped.yMin <= y && processParallelepiped.yMax >= y &&
        processParallelepiped.zMin <= z && processParallelepiped.zMax >= z)
    {
        return u[getLocalIndex(x, y, z, variables)];
    }

    int offset = 0;

    for (int index = 0; index < size; index++)
    {
        ProcessParallelepiped parallelepiped = recvNeighboursData[index];

        if (x < parallelepiped.xMin || x > parallelepiped.xMax ||
            y < parallelepiped.yMin || y > parallelepiped.yMax ||
            z < parallelepiped.zMin || z > parallelepiped.zMax)
        {
            offset += recvNeighboursData[index].size;
            continue;
        }

        return recv[offset + getIndex(x, y, z, parallelepiped)];
    }

    return -1;
}

// Оператор Лапласа
__host__ __device__ double calculateLaplaceOperator(int recvSize, const double *uData, int x, int y, int z, const double *uRecvData, const ProcessParallelepiped *recvNeighboursData, const SolverVariables &variables)
{
    GridSteps H = variables.H;
    double coeff = 2 * uData[getLocalIndex(x, y, z, variables)];

    double dx = (findValue(recvSize, uData, x - 1, y, z, uRecvData, recvNeighboursData, variables) + findValue(recvSize, uData, x + 1, y, z, uRecvData, recvNeighboursData, variables) - coeff) / (H.x * H.x);
    double dy = (findValue(recvSize, uData, x, y - 1, z, uRecvData, recvNeighboursData, variables) + findValue(recvSize, uData, x, y + 1, z, uRecvData, recvNeighboursData, variables) - coeff) / (H.y * H.y);
    double dz = (findValue(recvSize, uData, x, y, z - 1, uRecvData, recvNeighboursData, variables) + findValue(recvSize, uData, x, y, z + 1, uRecvData, recvNeighboursData, variables) - coeff) / (H.z * H.z);

    return dx + dy + dz;
}

// Получить значение граничного условия
__host__ __device__ double getBoundaryValue(int x, int y, int z, double t, const SolverVariables &variables)
{
    int N = variables.N;
    double hx = variables.H.x;
    double hy = variables.H.y;
    double hz = variables.H.z;

    // По X - первого рода
    if (x == 0 || x == N)
    {
        return 0;
    }

    // По Y - первого рода
    if (y == 0 || y == N)
    {
        return 0;
    }

    // По Z - периодическое значение
    if (z == 0 || z == N)
    {
        return getAnalyticValue(x * hx, y * hy, z * hz, t, variables.L);
    }

    return 0;
}

__global__ void fillBoundaryValuesX(double *u, int x, double tau, const SolverVariables &variables)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= processParallelepiped.dy * processParallelepiped.dz)
    {
        return;
    }

    int y = processParallelepiped.yMin + index / processParallelepiped.dz;
    int z = processParallelepiped.zMin + index % processParallelepiped.dz;

    u[getLocalIndex(x, y, z, variables)] = getBoundaryValue(x, y, z, tau, variables);
}

__global__ void fillBoundaryValuesY(double *u, int y, double tau, const SolverVariables &variables)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= processParallelepiped.dx * processParallelepiped.dz)
    {
        return;
    }

    int x = processParallelepiped.xMin + index / processParallelepiped.dz;
    int z = processParallelepiped.zMin + index % processParallelepiped.dz;

    u[getLocalIndex(x, y, z, variables)] = getBoundaryValue(x, y, z, tau, variables);
}

__global__ void fillBoundaryValuesZ(double *u, int z, double tau, const SolverVariables &variables)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= processParallelepiped.dx * processParallelepiped.dy)
    {
        return;
    }

    int x = processParallelepiped.xMin + index / processParallelepiped.dy;
    int y = processParallelepiped.yMin + index % processParallelepiped.dy;

    u[getLocalIndex(x, y, z, variables)] = getBoundaryValue(x, y, z, tau, variables);
}

void fillBoundaryValuesDefault(vector<double> &u, double tau, const SolverVariables &variables)
{
    int N = variables.N;

    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;

    // Граница при 0
    if (processParallelepiped.xMin == 0)
    {
        for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++)
        {
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++)
            {
                // u[индекс по x, y, z для текущего параллелепипеда]
                u[getLocalIndex(processParallelepiped.xMin, y, z, variables)] =
                    getBoundaryValue(processParallelepiped.xMin, y, z, tau, variables);
            }
        }
    }

    // Граница при N
    if (processParallelepiped.xMax == N)
    {
        for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++)
        {
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++)
            {
                u[getLocalIndex(processParallelepiped.xMax, y, z, variables)] =
                    getBoundaryValue(processParallelepiped.xMax, y, z, tau, variables);
            }
        }
    }

    // Граница при 0
    if (processParallelepiped.yMin == 0)
    {
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++)
        {
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++)
            {
                u[getLocalIndex(x, processParallelepiped.yMin, z, variables)] =
                    getBoundaryValue(x, processParallelepiped.yMin, z, tau, variables);
            }
        }
    }

    // Граница при N
    if (processParallelepiped.yMax == N)
    {
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++)
        {
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++)
            {
                u[getLocalIndex(x, processParallelepiped.yMax, z, variables)] =
                    getBoundaryValue(x, processParallelepiped.yMax, z, tau, variables);
            }
        }
    }

    // Граница при 0
    if (processParallelepiped.zMin == 0)
    {
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++)
        {
            for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++)
            {
                u[getLocalIndex(x, y, processParallelepiped.zMin, variables)] =
                    getBoundaryValue(x, y, processParallelepiped.zMin, tau, variables);
            }
        }
    }

    // Граница при N
    if (processParallelepiped.zMax == N)
    {
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++)
            for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++)
                u[getLocalIndex(x, y, processParallelepiped.zMax, variables)] =
                    getBoundaryValue(x, y, processParallelepiped.zMax, tau, variables);
    }
}

__global__ void fillZeroLayerKernel(double *u0, int xMin, int yMin, int zMin, const SolverVariables &variables)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;
    int index = blockIdx.x * blockDim.x * threadIdx.x;

    if (index >= processParallelepiped.size)
    {
        return;
    }

    int x = xMin + index / (processParallelepiped.dy * processParallelepiped.dz);
    int y = yMin + index % (processParallelepiped.dy * processParallelepiped.dz) / processParallelepiped.dz;
    int z = zMin + index % processParallelepiped.dz;

    u0[getLocalIndex(x, y, z, variables)] = getPhi(x * H.x, y * H.y, z * H.z, variables.L);
}

__global__ void fillFirstLayerKernel(double *u1, const double *u0, const double *u0Recv, const ProcessParallelepiped *recvNeighbours, int size, int xMin, int yMin, int zMin, const SolverVariables &variables)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;

    if (index >= processParallelepiped.size)
    {
        return;
    }

    int x = xMin + index / (processParallelepiped.dy * processParallelepiped.dz);
    int y = yMin + index % (processParallelepiped.dy * processParallelepiped.dz) / processParallelepiped.dz;
    int z = zMin + index % processParallelepiped.dz;

    u1[getLocalIndex(x, y, z, variables)] = u0[getLocalIndex(x, y, z, variables)] + variables.tau * variables.tau / 2 *
                                                                                        calculateLaplaceOperator(size, u0, x, y, z, u0Recv, recvNeighbours, variables);
}

__global__ void fillNextLayerKernel(double *u, const double *u0, const double *u1, const double *uRecv, const ProcessParallelepiped *recvNeighbors, int size, int xMin, int yMin, int zMin, const SolverVariables &variables)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;

    if (index > processParallelepiped.size)
    {
        return;
    }

    int x = xMin + index / (processParallelepiped.dy * processParallelepiped.dz);
    int y = yMin + index % (processParallelepiped.dy * processParallelepiped.dz) / processParallelepiped.dz;
    int z = zMin + index % processParallelepiped.dz;

    u[getLocalIndex(x, y, z, variables)] = 2 * u1[getLocalIndex(x, y, z, variables)] - u0[getLocalIndex(x, y, z, variables)] +
                                           variables.tau * variables.tau * calculateLaplaceOperator(size, u1, x, y, z, uRecv, recvNeighbors, variables);
}

template <typename T>
inline T *vectorToDevice(const vector<T> &vector, bool fill = true)
{
    T *deviceVector;
    hipMalloc((void **)&deviceVector, vector.size() * sizeof(T));

    if (fill)
    {
        hipMemcpyAsync(deviceVector, vector.data(), vector.size() * sizeof(T), hipMemcpyHostToDevice, 0);
    }

    return deviceVector;
}

void fillBoundaryValuesKernel(vector<double> &u, double tau, const SolverVariables &variables)
{
    int N = variables.N;
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    double *uDevice = vectorToDevice(u);

    if (processParallelepiped.xMin == 0)
    {
        fillBoundaryValuesX<<<((processParallelepiped.dy * processParallelepiped.dz + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE), GPU_BLOCK_SIZE>>>(uDevice, 0, tau, variables);
    }
    if (processParallelepiped.xMax == N)
    {
        fillBoundaryValuesX<<<((processParallelepiped.dy * processParallelepiped.dz + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE), GPU_BLOCK_SIZE>>>(uDevice, N, tau, variables);
    }
    if (processParallelepiped.yMin == 0)
    {
        fillBoundaryValuesY<<<((processParallelepiped.dx * processParallelepiped.dz + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE), GPU_BLOCK_SIZE>>>(uDevice, 0, tau, variables);
    }
    if (processParallelepiped.yMax == N)
    {
        fillBoundaryValuesY<<<((processParallelepiped.dx * processParallelepiped.dz + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE), GPU_BLOCK_SIZE>>>(uDevice, N, tau, variables);
    }
    if (processParallelepiped.zMin == 0)
    {
        fillBoundaryValuesZ<<<((processParallelepiped.dx * processParallelepiped.dy + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE), GPU_BLOCK_SIZE>>>(uDevice, 0, tau, variables);
    }
    if (processParallelepiped.zMax == N)
    {
        fillBoundaryValuesZ<<<((processParallelepiped.dx * processParallelepiped.dy + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE), GPU_BLOCK_SIZE>>>(uDevice, N, tau, variables);
    }

    hipMemcpyAsync(u.data(), uDevice, u.size() * sizeof(double), hipMemcpyDeviceToHost, 0);
    hipFree(uDevice);
}

// Заполнение граничными значениями
void fillBoundaryValues(vector<double> &u, double tau, const SolverVariables &variables)
{
    int N = variables.N;

    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;

    if (processParallelepiped.xMin > 0 && processParallelepiped.xMax < N &&
        processParallelepiped.yMin > 0 && processParallelepiped.yMax < N &&
        processParallelepiped.zMin > 0 && processParallelepiped.zMax < N)
    {
        return;
    }

    if (!USE_GPU)
    {
        fillBoundaryValuesDefault(u, tau, variables);
    }
    else
    {
        fillBoundaryValuesKernel(u, tau, variables);
    }
}

#pragma endregion

#pragma region data send recv

// Собрать параллелепипед в обособленный массив (вектор)
vector<double>
packParallelepiped(const vector<double> &u, const ProcessParallelepiped &parallelepiped,
                   const SolverVariables &variables)
{
    vector<double> packed(parallelepiped.size);

    for (int i = parallelepiped.xMin; i <= parallelepiped.xMax; i++)
    {
        for (int j = parallelepiped.yMin; j <= parallelepiped.yMax; j++)
        {
            for (int k = parallelepiped.zMin; k <= parallelepiped.zMax; k++)
            {
                packed[getIndex(i, j, k, parallelepiped)] = u[getLocalIndex(i, j, k, variables)];
            }
        }
    }

    return packed;
}

// Отправка и получение соседних значений
vector<double> sendRecvValues(const vector<double> &u, const SolverVariables &variables)
{
    unsigned long countOfNeighbours = variables.processIds.size();
    vector<double> recv(countOfNeighbours);
    int offset = 0;

    for (int i = 0; i < countOfNeighbours; i++)
    {
        vector<double> packed = packParallelepiped(u, variables.send[i], variables);
        recv.insert(recv.end(), variables.recv[i].size, 0);

        vector<MPI_Request> requests(2);
        vector<MPI_Status> statuses(2);

        // Отправляем буффер с начальным адресом packed.data() и кол-вом элементов variables.send[i].size типа double
        // в процесс с id variables.processIds[i] с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в requests[0]
        MPI_Isend(packed.data(), variables.send[i].size, MPI_DOUBLE, variables.processIds[i], 0, MPI_COMM_WORLD,
                  &requests[0]);
        // Получаем буффер с начальным адресом recv[i].data() и кол-вом элементов variables.recv[i].size типа double
        // из процесса с id variables.processIds[i] с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в requests[1]
        MPI_Irecv(recv.data(), variables.recv[i].size, MPI_DOUBLE, variables.processIds[i], 0,
                  MPI_COMM_WORLD, &requests[1]);
        // 2 = размер списка requests. Полученные статусы всех Isend/Irecv записываются в statuses
        // Ожидает выполнения всех MPI_Requests из списка requests
        MPI_Waitall(2, requests.data(), statuses.data());
        offset += variables.recv[i].size;
    }

    return recv;
}

// Отправка и получение общих значений
vector<double>
sendRecvTotal(const vector<double> &u, const vector<ProcessParallelepiped> &parallelepipeds,
              const SolverVariables &variables)
{
    if (variables.processId != MAIN_PROCESS_ID)
    {
        MPI_Request request;
        MPI_Status status;

        // Отправляем буффер с начальным адресом u.data() и кол-вом элементов variables.processParallelepiped.size типа double
        // в процесс с id MAIN_PROCESS_ID == 0 с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в request
        MPI_Isend(u.data(), variables.processParallelepiped.size, MPI_DOUBLE, MAIN_PROCESS_ID, 0, MPI_COMM_WORLD,
                  &request);
        // 1 = кол-во requests. Полученный статус всех Isend записывается в status
        // Ожидает выполнения всех MPI_Requests из списка requests
        MPI_Waitall(1, &request, &status);
        return u;
    }

    vector<double> uAll(variables.layerSize);
    ProcessParallelepiped parallelepipedAll = createParallelepiped(0, variables.N, 0, variables.N, 0, variables.N);

    for (int index = 0; index < variables.countOfProcesses; index++)
    {
        ProcessParallelepiped parallelepiped = parallelepipeds[index];
        vector<double> uI(parallelepiped.size);

        if (index == variables.processId)
        {
            uI = u;
        }
        else
        {
            vector<MPI_Request> requests(1);
            vector<MPI_Status> statuses(1);

            // Получаем буффер с начальным адресом uI.data() и кол-вом элементов parallelepiped.size типа double
            // из процесса с id index с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в requests[0]
            MPI_Irecv(uI.data(), parallelepiped.size, MPI_DOUBLE, index, 0, MPI_COMM_WORLD, &requests[0]);
            // 1 = кол-во requests. Полученный статус всех Irecv записывается в statuses
            // Ожидает выполнения всех MPI_Requests из списка requests
            MPI_Waitall(1, requests.data(), statuses.data());
        }

        for (int i = parallelepiped.xMin; i <= parallelepiped.xMax; i++)
        {
            for (int j = parallelepiped.yMin; j <= parallelepiped.yMax; j++)
            {
                for (int k = parallelepiped.zMin; k <= parallelepiped.zMax; k++)
                {
                    uAll[getIndex(i, j, k, parallelepipedAll)] = uI[getIndex(i, j, k, parallelepiped)];
                }
            }
        }
    }

    return uAll;
}

#pragma endregion

#pragma region Results fillers

void fillVectorByInitialValuesDefault(vector<vector<double>> &u, const SolverVariables &variables, int xMin, int xMax, int yMin, int yMax, int zMin, int zMax)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;
    for (int index = 0; index < processParallelepiped.size; index++)
    {
        int x = xMin + index / (processParallelepiped.dy * processParallelepiped.dz);
        int y = yMin + index % (processParallelepiped.dy * processParallelepiped.dz) / processParallelepiped.dz;
        int z = zMin + index % processParallelepiped.dz;

        u[0][getLocalIndex(x, y, z, variables)] = getPhi(x * H.x, y * H.y, z * H.z, variables.L);
    }

    vector<double> recv = sendRecvValues(u[0], variables);

    double coeff = variables.tau * variables.tau / 2;
    for (int index = 0; index < processParallelepiped.size; index++)
    {
        int x = xMin + index / (processParallelepiped.dy * processParallelepiped.dz);
        int y = yMin + index % (processParallelepiped.dy * processParallelepiped.dz) / processParallelepiped.dz;
        int z = zMin + index % processParallelepiped.dz;

        u[1][getLocalIndex(x, y, z, variables)] = u[0][getLocalIndex(x, y, z, variables)] + coeff * calculateLaplaceOperator(variables.recv.size(), u[0].data(), x, y, z, recv.data(), variables.recv.data(), variables);
    }
}

void fillVectorByInitialValuesKernel(vector<vector<double>> &u, const SolverVariables &variables, int xMin, int xMax, int yMin, int yMax, int zMin, int zMax)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    int nBlocks = (processParallelepiped.size + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE;

    double *u0Device = vectorToDevice(u[0], false);
    fillZeroLayerKernel<<<nBlocks, GPU_BLOCK_SIZE>>>(u0Device, xMin, yMin, zMin, variables);
    hipMemcpyAsync(u[0].data(), u0Device, u[0].size() * sizeof(double), hipMemcpyDeviceToHost, 0);

    vector<double> recv = sendRecvValues(u[0], variables);

    double *uRecvDevice = vectorToDevice(recv);
    double *u1Device = vectorToDevice(u[1], false);
    ProcessParallelepiped *recvNeighboursDevice = vectorToDevice(variables.recv);

    fillFirstLayerKernel<<<nBlocks, GPU_BLOCK_SIZE>>>(u1Device, u0Device, uRecvDevice, recvNeighboursDevice, variables.recv.size(), xMin, yMin, zMin, variables);
    hipMemcpyAsync(u[1].data(), u1Device, u[1].size() * sizeof(double), hipMemcpyDeviceToHost, 0);

    hipFree(u0Device);
    hipFree(uRecvDevice);
    hipFree(u1Device);
    hipFree(recvNeighboursDevice);
}

// Заполнение начальных условий
void fillVectorByInitialValues(vector<vector<double>> &u, const SolverVariables &variables)
{
    // Заполняем граничные условия;
    vector<double> &u0 = u[0];
    fillBoundaryValues(u0, 0, variables);

    vector<double> &u1 = u[1];
    fillBoundaryValues(u1, variables.tau, variables);

    ProcessParallelepiped target = variables.processParallelepiped;
    int N = variables.N;

    int xMin = max(target.xMin, 1);
    int xMax = min(target.xMax, N - 1);

    int yMin = max(target.yMin, 1);
    int yMax = min(target.yMax, N - 1);

    int zMin = max(target.zMin, 1);
    int zMax = min(target.zMax, N - 1);

    if (!USE_GPU)
    {
        fillVectorByInitialValuesDefault(u, variables, xMin, xMax, yMin, yMax, zMin, zMax);
    }
    else
    {
        fillVectorByInitialValuesKernel(u, variables, xMin, xMax, yMin, yMax, zMin, zMax);
    }
}

// Заполнение следующего слоя
void fillNextLayer(const vector<double> &u0, const vector<double> &u1, vector<double> &u, double t,
                   const SolverVariables &variables)
{
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    int N = variables.N;

    int xMin = max(processParallelepiped.xMin, 1);
    int xMax = min(processParallelepiped.xMax, N - 1);

    int yMin = max(processParallelepiped.yMin, 1);
    int yMax = min(processParallelepiped.yMax, N - 1);

    int zMin = max(processParallelepiped.zMin, 1);
    int zMax = min(processParallelepiped.zMax, N - 1);

    vector<double> recv = sendRecvValues(u1, variables);

    if (!USE_GPU)
    {
        for (int index = 0; index < processParallelepiped.size; index++)
        {
            int x = processParallelepiped.xMin + index / (processParallelepiped.dy * processParallelepiped.dz);
            int y = processParallelepiped.yMin + index % (processParallelepiped.dy * processParallelepiped.dz) / processParallelepiped.dz;
            int z = processParallelepiped.zMin + index % processParallelepiped.dz;

            u[getLocalIndex(x, y, z, variables)] = 2 * u1[getLocalIndex(x, y, z, variables)] - u0[getLocalIndex(x, y, z, variables)] +
                                                   variables.tau * variables.tau * calculateLaplaceOperator(variables.recv.size(), u1.data(), x, y, z, recv.data(), variables.recv.data(), variables);
        }
    }
    else
    {
        int nBlocks = (processParallelepiped.size + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE;

        double *u0Device = vectorToDevice(u0);
        double *u1Device = vectorToDevice(u1);
        double *uDevice = vectorToDevice(u, false);
        double *uRecvDevice = vectorToDevice(recv);
        ProcessParallelepiped *recvNeighborsDevice = vectorToDevice(variables.recv);

        fillNextLayerKernel<<<nBlocks, GPU_BLOCK_SIZE>>>(uDevice, u0Device, u1Device, uRecvDevice, recvNeighborsDevice, variables.recv.size(), xMin, yMin, zMin, variables);
        hipMemcpyAsync(u.data(), uDevice, u.size() * sizeof(double), hipMemcpyDeviceToHost, 0);

        hipFree(u0Device);
        hipFree(u1Device);
        hipFree(uDevice);
        hipFree(uRecvDevice);
        hipFree(recvNeighborsDevice);
    }

    fillBoundaryValues(u, t, variables);
}

// Заполнить аналитическими значениями
void fillAnalyticalValues(vector<double> &u, double t, const SolverVariables &variables)
{
    ProcessParallelepiped parallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;

    for (int x = parallelepiped.xMin; x <= parallelepiped.xMax; x++)
    {
        for (int y = parallelepiped.yMin; y <= parallelepiped.yMax; y++)
        {
            for (int z = parallelepiped.zMin; z <= parallelepiped.zMax; z++)
            {
                u[getLocalIndex(x, y, z, variables)] =
                    getAnalyticValue(x * H.x, y * H.y, z * H.z, t, variables.L);
            }
        }
    }
}

// Заполнить значениями с diff
void fillDifferenceValues(vector<double> &u, double t, const SolverVariables &variables)
{
    ProcessParallelepiped parallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;

    for (int x = parallelepiped.xMin; x <= parallelepiped.xMax; x++)
    {
        for (int y = parallelepiped.yMin; y <= parallelepiped.yMax; y++)
        {
            for (int z = parallelepiped.zMin; z <= parallelepiped.zMax; z++)
            {
                u[getLocalIndex(x, y, z, variables)] =
                    fabs(u[getLocalIndex(x, y, z, variables)]) -
                    getAnalyticValue(x * H.x, y * H.y, z * H.z, t, variables.L);
            }
        }
    }
}

__global__ void EvaluateErrorKernel(double *u, double t, double *error, const SolverVariables &variables)
{
    int idx = threadIdx.x;
    double max = 0;
    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;

    for (int index = idx; index < processParallelepiped.size; index++)
    {
        int x = processParallelepiped.xMin + index / (processParallelepiped.dy * processParallelepiped.dz);
        int y = processParallelepiped.yMin + index % (processParallelepiped.dy * processParallelepiped.dz) / processParallelepiped.dz;
        int z = processParallelepiped.zMin + index % processParallelepiped.dz;
        double delta = fabs(u[getLocalIndex(x, y, z, variables)] - getAnalyticValue(x * H.x, y * H.y, z * H.z, t, variables.L));

        if (delta > max)
        {
            max = delta;
        }
    }

    __shared__ double r[GPU_BLOCK_SIZE];

    r[idx] = max;
    __syncthreads();

    for (int size = GPU_BLOCK_SIZE / 2; size > 0; size /= 2)
    {
        if (idx < size)
        {
            r[idx] = r[idx] > r[idx + size] ? r[idx] : r[idx + size];
        }

        __syncthreads();
    }

    if (idx == 0)
    {
        *error = r[0];
    }
}

// Оценка погрешности на слое
double evaluateError(const vector<double> &u, double t, const SolverVariables &variables)
{
    double localError = 0;
    double error;
    ProcessParallelepiped parallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;

    if (!USE_GPU)
    {
        for (int x = parallelepiped.xMin; x <= parallelepiped.xMax; x++)
        {
            for (int y = parallelepiped.yMin; y <= parallelepiped.yMax; y++)
            {
                for (int z = parallelepiped.zMin; z <= parallelepiped.zMax; z++)
                {
                    localError = max(localError, fabs(u[getLocalIndex(x, y, z, variables)] -
                                                      getAnalyticValue(x * H.x, y * H.y, z * H.z, t, variables.L)));
                }
            }
        }
    }
    else
    {
        double *uDevice = vectorToDevice(u);
        double *errorDevice;
        hipMalloc((void **)&errorDevice, sizeof(double));

        EvaluateErrorKernel<<<1, GPU_BLOCK_SIZE>>>(uDevice, t, errorDevice, variables);

        hipMemcpyAsync(&localError, errorDevice, 1 * sizeof(double), hipMemcpyDeviceToHost, 0);
        hipFree(uDevice);
        hipFree(errorDevice);
    }

    // Отбираем максимальный localError в переменную error по всем процессам
    MPI_Reduce(&localError, &error, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

    return error;
}

// Cохранение слоя в формате json. Для построения графика на спец ресурсе
void saveValues(const vector<double> &u, double t, const vector<ProcessParallelepiped> &parallelepipeds,
                const char *filename,
                const SolverVariables &variables)
{
    vector<double> uAll = sendRecvTotal(u, parallelepipeds, variables);

    // Писать в файл может только мейн процесс (id == 0)
    if (variables.processId != MAIN_PROCESS_ID)
    {
        return;
    }

    ofstream f(filename);

    // Добавляем параметры запуска
    f << "{" << endl;
    f << "    \"Lx\": " << variables.L.x << ", " << endl;
    f << "    \"Ly\": " << variables.L.y << ", " << endl;
    f << "    \"Lz\": " << variables.L.z << ", " << endl;
    f << "    \"N\": " << variables.N << ", " << endl;
    f << "    \"t\": " << t << ", " << endl;
    f << "    \"u\": [" << endl;

    bool isFirstValuePrinted = false;

    // Печатаем слой
    for (int i = 0; i < variables.layerSize; i++)
    {
        if (isFirstValuePrinted)
        {
            f << ", " << endl;
        }
        else
        {
            isFirstValuePrinted = true;
        }

        f << "    " << uAll[i];
    }

    f << endl;
    f << "    ]" << endl;
    f << "}" << endl;

    f.close();
}

#pragma endregion

double makeSolution(SolverVariables &variables)
{
    int steps = variables.steps;
    int N = variables.N;
    double tau = variables.tau;

    // Разделяем параллелепипед по процессам на sub-параллелепипеды (Step 2)
    vector<ProcessParallelepiped> parallelepipeds = splitParallelepiped(N, variables.countOfProcesses);

    // Запоминаем рабочий параллелепипед
    variables.processParallelepiped = parallelepipeds[variables.processId];

    // Заполняем соседей
    fillNeighbours(parallelepipeds, variables);

    vector<vector<double>> u(3, vector<double>(variables.processParallelepiped.size));

    // Заполняем начальнные условия (step 4)
    fillVectorByInitialValues(u, variables);

    double error0 = evaluateError(u[0], 0, variables);
    double error1 = evaluateError(u[1], variables.tau, variables);

    if (variables.processId == MAIN_PROCESS_ID)
    {
        cout << "Layer 0 max error: " << error0 << endl;
        cout << "Layer 1 max error: " << error1 << endl;
    }

    double t = steps * tau;
    for (int step = 2; step <= steps; step++)
    {
        // Заполняем следующий слой u (step 5 + 6)
        fillNextLayer(u[(step + 1) % 3], u[(step + 2) % 3], u[step % 3], step * variables.tau, variables);

        // Вычисляем максимальную ошибку (step 8)
        double error = evaluateError(u[steps % 3], step * variables.tau, variables);
        if (variables.processId == MAIN_PROCESS_ID)
        {
            cout << "Layer " << step << " max error: " << error << endl;
        }
    }

    // for report.pdf (нарисовать графики)
    // saveValues(u[steps % 3], t, parallelepipeds, "numerical.json", variables);

    // fillDifferenceValues(u[steps % 3], t, variables);
    // saveValues(u[steps % 3], t, parallelepipeds, "difference.json", variables);

    // fillAnalyticalValues(u[0], t, variables);
    // saveValues(u[0], t, parallelepipeds, "analytical.json", variables);
    //

    return evaluateError(u[steps % 3], t, variables);
}

// ЛАЗАРЕВ В.А. / 628 группа / 2 вариант
int main(int argc, char *argv[])
{
    SolverVariables variables;

    int processId, countOfProcesses;
    // Инициализация MPI, создание группы процессов, создание области связи MPI_COMM_WORLD
    MPI_Init(NULL, NULL);
    // Определяем номер процесса (сохранится в переменную processId)
    MPI_Comm_rank(MPI_COMM_WORLD, &processId);
    // Определение числа процессов в области связи MPI_COMM_WORLD (сохранится в переменную countOfProcesses)
    MPI_Comm_size(MPI_COMM_WORLD, &countOfProcesses);

    initVariables(argc, argv, variables, processId, countOfProcesses);

    double start = MPI_Wtime();
    double error = 0;
    int loops = 1;

    for (size_t i = 0; i < loops; i++)
    {
        error += makeSolution(variables);
    }

    double end = MPI_Wtime();
    double diffTime = end - start;

    double minTime, maxTime, avgTime;

    // Отбираем минимальный diffTime по всем процессам
    MPI_Reduce(&diffTime, &minTime, 1, MPI_DOUBLE, MPI_MIN, MAIN_PROCESS_ID, MPI_COMM_WORLD);
    // Отбираем максимальный diffTime по всем процессам
    MPI_Reduce(&diffTime, &maxTime, 1, MPI_DOUBLE, MPI_MAX, MAIN_PROCESS_ID, MPI_COMM_WORLD);
    // Отбираем средний diffTime по всем процессам
    MPI_Reduce(&diffTime, &avgTime, 1, MPI_DOUBLE, MPI_SUM, MAIN_PROCESS_ID, MPI_COMM_WORLD);

    avgTime = avgTime / countOfProcesses;

    // Чтоб печатал информацию только один процесс
    if (processId == MAIN_PROCESS_ID)
    {
        ofstream fout("results.txt");
        fout << "### Lx = Ly = Lz = " << variables.L.x << ", N = " << variables.N << ", K = " << variables.K << endl
             << endl;
        fout << "| Число MPI процессов (P) | Время решения (с) | Ускорение | Погрешность |" << endl;
        fout << "|                     :-: |               :-: |       :-: |         :-: |" << endl;

        fout << "| " << setw(23) << variables.countOfProcesses;
        fout << " | " << setw(17) << (maxTime / loops);
        fout << " | "
             << "         ";
        fout << " | " << setw(11) << (error / loops);
        fout << " |" << endl;

        fout << "Final error: " << error / loops << endl;
        fout << "Minimal time (s): " << minTime / loops << endl;
        fout << "Maximum time (s): " << maxTime / loops << endl;
        fout << "Average time (s): " << avgTime / loops << endl
             << endl;
        fout.close();
    }

    MPI_Finalize();

    return 0;
}