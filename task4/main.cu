#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <mpi/mpi.h>
//#include <mpi.h>
#include <string>
#include <cstring>
#include <cmath>
#include <vector>
#include <omp.h>

using namespace std;

#pragma region Entities and constants
const int MAIN_PROCESS_ID = 0;

// Оси разбиения
enum Axis {
    X, Y, Z,
};

// Исходный параллелпипед
struct Parallelepiped {
    // Размеры параллелепипеда по каждой из осей
    double x;
    double y;
    double z;
};

// Параллелпипед, обрабтываемый отдельным процессом
struct ProcessParallelepiped {
    // Границы по оси X
    int xMin, xMax;

    // Границы по оси Y
    int yMin, yMax;

    // Границы по оси Z
    int zMin, zMax;

    // Размеры по каждой из осей + объем области
    int dx, dy, dz, size;
};

// Шаги пространственной сетки по каждой из осей
struct GridSteps {
    double x;
    double y;
    double z;
};

struct SolverVariables {
    // Исходный параллелпипед
    Parallelepiped L;
    // T - итоговое время
    double T;
    // N - количество точек пространственной сетки / K - количество точек временной сетки / steps - кол-во шагов для решения
    int N, K, steps;

    // Шаги пространственной сетки по каждой из осей
    GridSteps H;
    // Шаг временной сетки
    double tau;
    // Размер слоя
    int layerSize;

    // Количество нитей OMP
    int ompThreadsCount;

    // Id процесса / количество MPI-процессов
    int processId, countOfProcesses;

    // Параллелпипед, обрабтываемый отдельным процессом
    ProcessParallelepiped processParallelepiped;
    // Параллелепипеды-соседи на передачу
    vector<ProcessParallelepiped> send;
    // Параллелепипеды-соседи на прием
    vector<ProcessParallelepiped> recv;
    // Параллелепипеды-соседи в формате id процесса
    vector<int> processIds;

};
#pragma endregion

#pragma region Handle and init arguments

float getFloatValueFromArg(const char *key, int argc, char *argv[], float defaultValue) {
    float value = 0;
    bool valueFound = false;
    // Первый параметр - ссылка на сборку
    for (int i = 1; i < argc; i++) {
        string currentArgument(argv[i]);
        int argName = currentArgument.find(key);
        if (argName != string::npos) {
            string argString = currentArgument.substr(argName + strlen(key));
            try {
                value = atof(argString.c_str());
                valueFound = true;
                break;
            } catch (...) {
                // Не поддерживается на кластерах
                // throw runtime_error("Invalid input epsilon");
                throw;
            }
        }
    }

    return valueFound ? value : defaultValue;
}

int getIntValueFromArg(const char *key, int argc, char *argv[], int defaultValue) {
    int value = 0;
    bool valueFound = false;
    // Первый параметр - ссылка на сборку
    for (int i = 1; i < argc; i++) {
        string currentArgument(argv[i]);
        int argName = currentArgument.find(key);
        if (argName != string::npos) {
            string argString = currentArgument.substr(argName + strlen(key));
            try {
                value = atoi(argString.c_str());
                valueFound = true;
                break;
            } catch (...) {
                // Не поддерживается на кластерах
                // throw runtime_error("Invalid input epsilon");
                throw;
            }
        }
    }

    return valueFound ? value : defaultValue;
}

void initVariables(int argc, char *argv[], SolverVariables &variables, int processId, int countOfProcesses) {
    // Первый параметр - ссылка на сборку
    variables.L.x = getFloatValueFromArg("-Lx=", argc, argv, 1);
    variables.L.y = getFloatValueFromArg("-Ly=", argc, argv, 1);
    variables.L.z = getFloatValueFromArg("-Lz=", argc, argv, 1);
    variables.T = getFloatValueFromArg("-T=", argc, argv, 1);
    variables.N = getIntValueFromArg("-N=", argc, argv, 128);
    variables.K = getIntValueFromArg("-K=", argc, argv, 2000);
    variables.steps = getIntValueFromArg("-steps=", argc, argv, 5);
    variables.ompThreadsCount = getIntValueFromArg("-omp=", argc, argv, 1);
    // Остальные параметры будут игнорироваться (ну или позже добавлю какие-нибудь свои кастомные)

    variables.H.x = variables.L.x / variables.N;
    variables.H.y = variables.L.y / variables.N;
    variables.H.z = variables.L.z / variables.N;
    variables.tau = variables.T / variables.K;
    variables.layerSize = pow(variables.N + 1, 3);

    variables.processId = processId;
    variables.countOfProcesses = countOfProcesses;
}

#pragma endregion

#pragma region Split parallelepiped

// Конструктор параллелепипеда по макс/мин координатам
ProcessParallelepiped createParallelepiped(int xMin, int xMax, int yMin, int yMax, int zMin, int zMax) {
    int dx = xMax - xMin + 1;
    int dy = yMax - yMin + 1;
    int dz = zMax - zMin + 1;
    int size = dx * dy * dz;

    return ProcessParallelepiped{xMin, xMax, yMin, yMax, zMin, zMax, dx, dy, dz, size};
}

// Блочное разбиение параллелепипеда
void makeSplit(int xMin, int xMax, int yMin, int yMax, int zMin, int zMax, int countOfProcesses, Axis axis,
               vector<ProcessParallelepiped> &subParallelepipeds) {

    // Если один процесс, то ему достается полный параллелепипед
    if (countOfProcesses == 1) {
        subParallelepipeds.push_back(createParallelepiped(xMin, xMax, yMin, yMax, zMin, zMax));
        return;
    }

    // Делаем область по текущей оси, делаем паралл, разбиваем дальше
    if (countOfProcesses % 2 == 1) {
        if (axis == X) {
            int newXMax = xMin + (xMax - xMin) / countOfProcesses;
            subParallelepipeds.push_back(createParallelepiped(xMin, newXMax, yMin, yMax, zMin, zMax));

            xMin = newXMax + 1;
            axis = Y;
        } else if (axis == Y) {
            int newYMax = yMin + (yMax - yMin) / countOfProcesses;
            subParallelepipeds.push_back(createParallelepiped(xMin, xMax, yMin, newYMax, zMin, zMax));

            yMin = newYMax + 1;
            axis = Z;
        } else if (axis == Z) {
            int newZMax = zMin + (zMax - zMin) / countOfProcesses;
            subParallelepipeds.push_back(createParallelepiped(xMin, xMax, yMin, yMax, zMin, newZMax));

            zMin = newZMax + 1;
            axis = X;
        }

        countOfProcesses--;
    }

    // Для оси делим область пополам и запускаем рекурсию для sub-параллелепипедов
    int newCountOfProcesses = countOfProcesses / 2;
    if (axis == X) {
        int newXMax = (xMin + xMax) / 2;
        makeSplit(xMin, newXMax, yMin, yMax, zMin, zMax, newCountOfProcesses, Y, subParallelepipeds);
        makeSplit(newXMax + 1, xMax, yMin, yMax, zMin, zMax, newCountOfProcesses, Y, subParallelepipeds);
    } else if (axis == Y) {
        int newYMax = (yMin + yMax) / 2;
        makeSplit(xMin, xMax, yMin, newYMax, zMin, zMax, newCountOfProcesses, Z, subParallelepipeds);
        makeSplit(xMin, xMax, newYMax + 1, yMax, zMin, zMax, newCountOfProcesses, Z, subParallelepipeds);
    } else if (axis == Z) {
        int newZMax = (zMin + zMax) / 2;
        makeSplit(xMin, xMax, yMin, yMax, zMin, newZMax, newCountOfProcesses, X, subParallelepipeds);
        makeSplit(xMin, xMax, yMin, yMax, newZMax + 1, zMax, newCountOfProcesses, X, subParallelepipeds);
    }

}

// Запуск разбиения параллелпипеда на более мелкие
vector<ProcessParallelepiped> splitParallelepiped(int N, int countOfProcesses) {
    vector<ProcessParallelepiped> subParallelepipeds;

    // Коэффициенты - параметры исходного параллелепипеда
    makeSplit(0, N, 0, N, 0, N, countOfProcesses, X, subParallelepipeds);

    return subParallelepipeds;
}

#pragma endregion

#pragma region Actions with neighbours

// Проверка на то, что первый параллелепипед (координаты с 1) находится внутри второго (координаты с 2)
bool isInside(int xMin1, int xMax1, int yMin1, int yMax1, int xMin2, int xMax2, int yMin2, int yMax2) {
    return xMin2 <= xMin1 && xMax1 <= xMax2 && yMin2 <= yMin1 && yMax1 <= yMax2;
}

// Получить соприкосновение двух параллелипедов (true - если соприкосновение есть, иначе - false). result - соприкосновение
bool
getNeighbours(const ProcessParallelepiped &first, const ProcessParallelepiped &second, ProcessParallelepiped &result) {
    // Если начало одного == конец второго по оси X
    if (first.xMin == second.xMax + 1 || second.xMin == first.xMax + 1) {
        // Берем точку соприкосновения двух параллелепипедов
        int x = (first.xMin == second.xMax + 1) ? first.xMin : first.xMax;

        // Если fisrt внутри second
        if (isInside(first.yMin, first.yMax, first.zMin, first.zMax, second.yMin, second.yMax, second.zMin,
                     second.zMax)) {
            // Создадим параллепипед по X размером 1 (соприкосновение блоков) и размерами first => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(x, x, first.yMin, first.yMax, first.zMin, first.zMax);
            return true;
        }

        // Если second внутри first
        if (isInside(second.yMin, second.yMax, second.zMin, second.zMax, first.yMin, first.yMax, first.zMin,
                     first.zMax)) {
            // Создадим параллепипед по X размером 1 (соприкосновение блоков) и размерами second => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(x, x, second.yMin, second.yMax, second.zMin, second.zMax);
            return true;
        }

        return false;
    }

    // Если начало одного == конец второго по оси Y
    if (first.yMin == second.yMax + 1 || second.yMin == first.yMax + 1) {
        // Берем точку соприкосновения двух параллелепипедов
        int y = (first.yMin == second.yMax + 1) ? first.yMin : first.yMax;

        // Если fisrt внутри second
        if (isInside(first.xMin, first.xMax, first.zMin, first.zMax, second.xMin, second.xMax, second.zMin,
                     second.zMax)) {
            // Создадим параллепипед по Y размером 1 (соприкосновение блоков) и размерами first => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(first.xMin, first.xMax, y, y, first.zMin, first.zMax);
            return true;
        }

        // Если second внутри first
        if (isInside(second.xMin, second.xMax, second.zMin, second.zMax, first.xMin, first.xMax, first.zMin,
                     first.zMax)) {
            // Создадим параллепипед по Y размером 1 (соприкосновение блоков) и размерами second => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(second.xMin, second.xMax, y, y, second.zMin, second.zMax);
            return true;
        }

        return false;
    }

    // Если начало одного == конец второго по оси Z
    if (first.zMin == second.zMax + 1 || second.zMin == first.zMax + 1) {
        // Берем точку соприкосновения двух параллелепипедов
        int z = (first.zMin == second.zMax + 1) ? first.zMin : first.zMax;

        // Если fisrt внутри second
        if (isInside(first.xMin, first.xMax, first.yMin, first.yMax, second.xMin, second.xMax, second.yMin,
                     second.yMax)) {
            // Создадим параллепипед по Z размером 1 (соприкосновение блоков) и размерами first => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(first.xMin, first.xMax, first.yMin, first.yMax, z, z);
            return true;
        }

        // Если second внутри first
        if (isInside(second.xMin, second.xMax, second.yMin, second.yMax, first.xMin, first.xMax, first.yMin,
                     first.yMax)) {
            // Создадим параллепипед по Z размером 1 (соприкосновение блоков) и размерами second => result - параллелепипед, от которого получать/посылать информацию
            result = createParallelepiped(second.xMin, second.xMax, second.yMin, second.yMax, z, z);
            return true;
        }

        return false;
    }

    return false;
}

// Заполняем соседей
void fillNeighbours(vector<ProcessParallelepiped> &parallelepipeds, SolverVariables &variables) {
    variables.send.clear();
    variables.recv.clear();
    variables.processIds.clear();
    ProcessParallelepiped targetParallelepiped = variables.processParallelepiped;

    for (int i = 0; i < variables.countOfProcesses; i++) {
        // Самого себя не обрабатываем
        if (i == variables.processId) {
            continue;
        }

        ProcessParallelepiped send;
        ProcessParallelepiped recv;

        ProcessParallelepiped processParallelepiped = parallelepipeds[i];
        // Получаем соседей, которым будем отправлять информацию
        if (!getNeighbours(targetParallelepiped, processParallelepiped, send)) {
            continue;
        }

        // Получаем соседей, от которых будем получать информацию
        getNeighbours(processParallelepiped, targetParallelepiped, recv);
        variables.processIds.push_back(i);
        variables.send.push_back(send);
        variables.recv.push_back(recv);
    }
}

#pragma endregion

#pragma region Math calculations

// Аналитическое решение
double getAnalyticValue(double x, double y, double z, double t, const Parallelepiped &L) {
    double at = M_PI * sqrt(1 / pow(L.x, 2) + 1 / pow(L.y, 2) + 4 / pow(L.z, 2));

    return sin(M_PI * x / L.x) * sin(M_PI * y / L.y) * sin(2 * z * M_PI / L.z) * cos(at * t + 2 * M_PI);
}

// Начальные условия
double getPhi(double x, double y, double z, const Parallelepiped &L) {
    return getAnalyticValue(x, y, z, 0, L);
}

// Получить индекс по x, y, z для конкрентного параллелепипеда
int getIndex(int x, int y, int z, const ProcessParallelepiped &target) {
    return (x - target.xMin) * target.dy * target.dz + (y - target.yMin) * target.dz + (z - target.zMin);
}

// Получить локальный индекс
int getLocalIndex(int x, int y, int z, const SolverVariables &variables) {
    return getIndex(x, y, z, variables.processParallelepiped);
}

double findValue(const vector<double> &u, int x, int y, int z, const vector<vector<double>> &recv,
                 const SolverVariables &variables) {
    for (int index = 0; index < variables.processIds.size(); index++) {
        ProcessParallelepiped parallelepiped = variables.recv[index];

        if (x < parallelepiped.xMin || x > parallelepiped.xMax ||
            y < parallelepiped.yMin || y > parallelepiped.yMax ||
            z < parallelepiped.zMin || z > parallelepiped.zMax) {
            continue;
        }

        return recv[index][getIndex(x, y, z, parallelepiped)];
    }

    return u[getLocalIndex(x, y, z, variables)];
}

// Оператор Лапласа
double calculateLaplaceOperator(const vector<double> &u, int x, int y, int z, const vector<vector<double>> &recv,
                                const SolverVariables &variables, double localUValue) {
    GridSteps H = variables.H;
//    double start = MPI_Wtime();
    double coeff = 2 * localUValue;
//    double startTemp = MPI_Wtime();
//    double tempDx = findValue(u, x - 1, y, z, recv, variables) + findValue(u, x + 1, y, z, recv, variables) -
//                    coeff;
//    double endTemp = MPI_Wtime();
//        if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "tempDx time: " << endTemp - startTemp << endl;
//    }
    double dx = (findValue(u, x - 1, y, z, recv, variables) + findValue(u, x + 1, y, z, recv, variables) -
                 coeff) / (H.x * H.x);
    double dy = (findValue(u, x, y - 1, z, recv, variables) + findValue(u, x, y + 1, z, recv, variables) -
                 coeff) / (H.y * H.y);
    double dz = (findValue(u, x, y, z - 1, recv, variables) + findValue(u, x, y, z + 1, recv, variables) -
                 coeff) / (H.z * H.z);
//    double end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "Laplace time: " << end - start << endl;
//    }

    return dx + dy + dz;
}

// Получить значение граничного условия
double getBoundaryValue(int x, int y, int z, double t, const SolverVariables &variables) {
    int N = variables.N;
    double hx = variables.H.x;
    double hy = variables.H.y;
    double hz = variables.H.z;

    // По X - первого рода
    if (x == 0 || x == N) {
        return 0;
    }

    // По Y - первого рода
    if (y == 0 || y == N) {
        return 0;
    }

    // По Z - периодическое значение
    if (z == 0 || z == N) {
        return getAnalyticValue(x * hx, y * hy, z * hz, t, variables.L);
    }

    return 0;
}

// Заполнение граничными значениями
void fillBoundaryValues(vector<double> &u, double tau, const SolverVariables &variables) {
    int N = variables.N;

    ProcessParallelepiped processParallelepiped = variables.processParallelepiped;

    // Граница при 0
    if (processParallelepiped.xMin == 0) {
        // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
        for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++) {
#pragma omp parallel for
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++) {
                // u[индекс по x, y, z для текущего параллелепипеда]
                u[getLocalIndex(processParallelepiped.xMin, y, z, variables)] =
                        getBoundaryValue(processParallelepiped.xMin, y, z, tau, variables);
            }
        }
    }

    // Граница при N
    if (processParallelepiped.xMax == N) {
        // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
        for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++) {
#pragma omp parallel for
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++) {
                u[getLocalIndex(processParallelepiped.xMax, y, z, variables)] =
                        getBoundaryValue(processParallelepiped.xMax, y, z, tau, variables);
            }
        }
    }

    // Граница при 0
    if (processParallelepiped.yMin == 0) {
        // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++) {
#pragma omp parallel for
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++) {
                u[getLocalIndex(x, processParallelepiped.yMin, z, variables)] =
                        getBoundaryValue(x, processParallelepiped.yMin, z, tau, variables);
            }
        }
    }

    // Граница при N
    if (processParallelepiped.yMax == N) {
        // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++) {
#pragma omp parallel for
            for (int z = processParallelepiped.zMin; z <= processParallelepiped.zMax; z++) {
                u[getLocalIndex(x, processParallelepiped.yMax, z, variables)] =
                        getBoundaryValue(x, processParallelepiped.yMax, z, tau, variables);
            }
        }
    }

    // Граница при 0
    if (processParallelepiped.zMin == 0) {
        // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++) {
#pragma omp parallel for
            for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++) {
                u[getLocalIndex(x, y, processParallelepiped.zMin, variables)] =
                        getBoundaryValue(x, y, processParallelepiped.zMin, tau, variables);
            }
        }
    }

    // Граница при N
    if (processParallelepiped.zMax == N) {
        // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
        for (int x = processParallelepiped.xMin; x <= processParallelepiped.xMax; x++)
#pragma omp parallel for
                for (int y = processParallelepiped.yMin; y <= processParallelepiped.yMax; y++)
                    u[getLocalIndex(x, y, processParallelepiped.zMax, variables)] =
                            getBoundaryValue(x, y, processParallelepiped.zMax, tau, variables);
    }
}

#pragma endregion

#pragma region data send recv

// Собрать параллелепипед в обособленный массив (вектор)
vector<double>
packParallelepiped(const vector<double> &u, const ProcessParallelepiped &parallelepiped,
                   const SolverVariables &variables) {
    vector<double> packed(parallelepiped.size);

    // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
    for (int i = parallelepiped.xMin; i <= parallelepiped.xMax; i++) {
#pragma omp parallel for
        for (int j = parallelepiped.yMin; j <= parallelepiped.yMax; j++) {
#pragma omp parallel for
            for (int k = parallelepiped.zMin; k <= parallelepiped.zMax; k++) {
                packed[getIndex(i, j, k, parallelepiped)] = u[getLocalIndex(i, j, k, variables)];
            }
        }
    }

    return packed;
}

// Отправка и получение соседних значений
vector<vector<double>> sendRecvValues(const vector<double> &u, const SolverVariables &variables) {
    unsigned long countOfNeighbours = variables.processIds.size();
    vector<vector<double>> recv(countOfNeighbours);

    for (int i = 0; i < countOfNeighbours; i++) {
        vector<double> packed = packParallelepiped(u, variables.send[i], variables);
        recv[i] = vector<double>(variables.recv[i].size);

        vector<MPI_Request> requests(2);
        vector<MPI_Status> statuses(2);

        // Отправляем буффер с начальным адресом packed.data() и кол-вом элементов variables.send[i].size типа double
        // в процесс с id variables.processIds[i] с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в requests[0]
        MPI_Isend(packed.data(), variables.send[i].size, MPI_DOUBLE, variables.processIds[i], 0, MPI_COMM_WORLD,
                  &requests[0]);
        // Получаем буффер с начальным адресом recv[i].data() и кол-вом элементов variables.recv[i].size типа double
        // из процесса с id variables.processIds[i] с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в requests[1]
        MPI_Irecv(recv[i].data(), variables.recv[i].size, MPI_DOUBLE, variables.processIds[i], 0,
                  MPI_COMM_WORLD, &requests[1]);
        // 2 = размер списка requests. Полученные статусы всех Isend/Irecv записываются в statuses
        // Ожидает выполнения всех MPI_Requests из списка requests
        MPI_Waitall(2, requests.data(), statuses.data());
    }

    return recv;
}

// Отправка и получение общих значений
vector<double>
sendRecvTotal(const vector<double> &u, const vector<ProcessParallelepiped> &parallelepipeds,
              const SolverVariables &variables) {
    if (variables.processId != MAIN_PROCESS_ID) {
        MPI_Request request;
        MPI_Status status;

        // Отправляем буффер с начальным адресом u.data() и кол-вом элементов variables.processParallelepiped.size типа double
        // в процесс с id MAIN_PROCESS_ID == 0 с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в request
        MPI_Isend(u.data(), variables.processParallelepiped.size, MPI_DOUBLE, MAIN_PROCESS_ID, 0, MPI_COMM_WORLD,
                  &request);
        // 1 = кол-во requests. Полученный статус всех Isend записывается в status
        // Ожидает выполнения всех MPI_Requests из списка requests
        MPI_Waitall(1, &request, &status);
        return u;
    }

    vector<double> uAll(variables.layerSize);
    ProcessParallelepiped parallelepipedAll = createParallelepiped(0, variables.N, 0, variables.N, 0, variables.N);

    for (int index = 0; index < variables.countOfProcesses; index++) {
        ProcessParallelepiped parallelepiped = parallelepipeds[index];
        vector<double> uI(parallelepiped.size);

        if (index == variables.processId) {
            uI = u;
        } else {
            vector<MPI_Request> requests(1);
            vector<MPI_Status> statuses(1);

            // Получаем буффер с начальным адресом uI.data() и кол-вом элементов parallelepiped.size типа double
            // из процесса с id index с пометкой 0 в коммутаторе MPI_COMM_WORLD. Выходное значение записывается в requests[0]
            MPI_Irecv(uI.data(), parallelepiped.size, MPI_DOUBLE, index, 0, MPI_COMM_WORLD, &requests[0]);
            // 1 = кол-во requests. Полученный статус всех Irecv записывается в statuses
            // Ожидает выполнения всех MPI_Requests из списка requests
            MPI_Waitall(1, requests.data(), statuses.data());
        }

        for (int i = parallelepiped.xMin; i <= parallelepiped.xMax; i++) {
            for (int j = parallelepiped.yMin; j <= parallelepiped.yMax; j++) {
                for (int k = parallelepiped.zMin; k <= parallelepiped.zMax; k++) {
                    uAll[getIndex(i, j, k, parallelepipedAll)] = uI[getIndex(i, j, k, parallelepiped)];
                }
            }
        }
    }

    return uAll;
}

#pragma endregion

#pragma region Results fillers

// Заполнение начальных условий
void fillVectorByInitialValues(vector<vector<double>> &u, const SolverVariables &variables) {
    // Заполняем граничные условия
//    double start = MPI_Wtime();
    vector<double> &u0 = u[0];
    fillBoundaryValues(u0, 0, variables);
    vector<double> &u1 = u[1];
    fillBoundaryValues(u1, variables.tau, variables);
//    double end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "fillBoundaryValues time:" << end - start << endl;
//    }

    ProcessParallelepiped target = variables.processParallelepiped;
    int N = variables.N;

    int xMin = max(target.xMin, 1);
    int xMax = min(target.xMax, N - 1);

    int yMin = max(target.yMin, 1);
    int yMax = min(target.yMax, N - 1);

    int zMin = max(target.zMin, 1);
    int zMax = min(target.zMax, N - 1);

    GridSteps H = variables.H;

//    start = MPI_Wtime();
    // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
    for (int x = xMin; x <= xMax; x++) {
#pragma omp parallel for
        for (int y = yMin; y <= yMax; y++) {
#pragma omp parallel for
            for (int z = zMin; z <= zMax; z++) {
                u0[getLocalIndex(x, y, z, variables)] = getPhi(x * H.x, y * H.y, z * H.z, variables.L);
            }
        }
    }
//    end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "getPhi time:" << end - start << endl;
//    }

//    start = MPI_Wtime();
    vector<vector<double>> recv = sendRecvValues(u0, variables);
//    end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "sendRecvValues time:" << end - start << endl;
//    }

//    start = MPI_Wtime();
    // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "loop size " << (xMax - xMin + 1) * (yMax - yMin + 1) * (zMax - zMin + 1) << endl;
//    }
    double coeff = variables.tau * variables.tau / 2;
#pragma omp parallel for
    for (int x = xMin; x <= xMax; x++) {
#pragma omp parallel for
        for (int y = yMin; y <= yMax; y++) {
#pragma omp parallel for
            for (int z = zMin; z <= zMax; z++) {
                double &currentValueU0 = u0[getLocalIndex(x, y, z, variables)];

                u1[getLocalIndex(x, y, z, variables)] =
                        currentValueU0 + coeff * calculateLaplaceOperator(u0, x, y, z, recv,
                                                                          variables, currentValueU0);
            }
        }
    }
//    end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "calculateLaplaceOperator time:" << end - start << endl;
//    }
}

// Заполнение следующего слоя
void fillNextLayer(const vector<double> &u0, const vector<double> &u1, vector<double> &u, double t,
                   const SolverVariables &variables) {
    int xMin = max(variables.processParallelepiped.xMin, 1);
    int xMax = min(variables.processParallelepiped.xMax, variables.N - 1);

    int yMin = max(variables.processParallelepiped.yMin, 1);
    int yMax = min(variables.processParallelepiped.yMax, variables.N - 1);

    int zMin = max(variables.processParallelepiped.zMin, 1);
    int zMax = min(variables.processParallelepiped.zMax, variables.N - 1);

    vector<vector<double>> recv = sendRecvValues(u1, variables);

    double tauSquare = variables.tau * variables.tau;
    // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
    for (int x = xMin; x <= xMax; x++) {
#pragma omp parallel for
        for (int y = yMin; y <= yMax; y++) {
#pragma omp parallel for
            for (int z = zMin; z <= zMax; z++) {
                double u1Value = u1[getLocalIndex(x, y, z, variables)];

                u[getLocalIndex(x, y, z, variables)] =
                        2 * u1Value -
                        u0[getLocalIndex(x, y, z, variables)] +
                        tauSquare * calculateLaplaceOperator(u1, x, y, z, recv, variables, u1Value);
            }
        }
    }

    fillBoundaryValues(u, t, variables);
}

// Заполнить аналитическими значениями
void fillAnalyticalValues(vector<double> &u, double t, const SolverVariables &variables) {
    ProcessParallelepiped parallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;

    // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
    for (int x = parallelepiped.xMin; x <= parallelepiped.xMax; x++) {
#pragma omp parallel for
        for (int y = parallelepiped.yMin; y <= parallelepiped.yMax; y++) {
#pragma omp parallel for
            for (int z = parallelepiped.zMin; z <= parallelepiped.zMax; z++) {
                u[getLocalIndex(x, y, z, variables)] =
                        getAnalyticValue(x * H.x, y * H.y, z * H.z, t, variables.L);
            }
        }
    }
}

// Заполнить значениями с diff
void fillDifferenceValues(vector<double> &u, double t, const SolverVariables &variables) {
    ProcessParallelepiped parallelepiped = variables.processParallelepiped;
    GridSteps H = variables.H;

    // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
#pragma omp parallel for
    for (int x = parallelepiped.xMin; x <= parallelepiped.xMax; x++) {
#pragma omp parallel for
        for (int y = parallelepiped.yMin; y <= parallelepiped.yMax; y++) {
#pragma omp parallel for
            for (int z = parallelepiped.zMin; z <= parallelepiped.zMax; z++) {
                u[getLocalIndex(x, y, z, variables)] =
                        fabs(u[getLocalIndex(x, y, z, variables)]) -
                        getAnalyticValue(x * H.x, y * H.y, z * H.z, t, variables.L);
            }
        }
    }
}

// Оценка погрешности на слое
double evaluateError(const vector<double> &u, double t, const SolverVariables &variables) {
    double localError = 0, error = 0;
    int N = variables.N;
    GridSteps H = variables.H;
    ProcessParallelepiped parallelepiped = variables.processParallelepiped;

    // https://pages.tacc.utexas.edu/~eijkhout/pcse/html/omp-reduction.html
    // Выполняется max в переменной localError (как альтернатива можно было б использовать critical секцию)
    // Директива указывает на то, что данный цикл следует разделить по итерациям между потоками.
// #pragma omp parallel reduction(max: localError)
#pragma omp parallel for
    for (int x = parallelepiped.xMin; x <= parallelepiped.xMax; x++) {
#pragma omp parallel for
        for (int y = parallelepiped.yMin; y <= parallelepiped.yMax; y++) {
#pragma omp parallel for
            for (int z = parallelepiped.zMin; z <= parallelepiped.zMax; z++) {
#pragma omp critical
                {
                    localError = max(localError, fabs(u[getLocalIndex(x, y, z, variables)] -
                                                      getAnalyticValue(x * H.x, y * H.y, z * H.z, t, variables.L)));
                }
            }
        }
    }

    // Отбираем максимальный localError в переменную error по всем процессам
    MPI_Reduce(&localError, &error, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

    return error;
}

// Cохранение слоя в формате json. Для построения графика на спец ресурсе
void
saveValues(const vector<double> &u, double t, const vector<ProcessParallelepiped> &parallelepipeds, const char *filename,
           const SolverVariables &variables) {
    vector<double> uAll = sendRecvTotal(u, parallelepipeds, variables);

    // Писать в файл может только мейн процесс (id == 0)
    if (variables.processId != MAIN_PROCESS_ID) {
        return;
    }

    ofstream f(filename);

    // Добавляем параметры запуска
    f << "{" << endl;
    f << "    \"Lx\": " << variables.L.x << ", " << endl;
    f << "    \"Ly\": " << variables.L.y << ", " << endl;
    f << "    \"Lz\": " << variables.L.z << ", " << endl;
    f << "    \"N\": " << variables.N << ", " << endl;
    f << "    \"t\": " << t << ", " << endl;
    f << "    \"u\": [" << endl;

    bool isFirstValuePrinted = false;

    // Печатаем слой
    for (int i = 0; i < variables.layerSize; i++) {
        if (isFirstValuePrinted) {
            f << ", " << endl;
        } else {
            isFirstValuePrinted = true;
        }

        f << "    " << uAll[i];
    }

    f << endl;
    f << "    ]" << endl;
    f << "}" << endl;

    f.close();
}

#pragma endregion

double makeSolution(SolverVariables &variables) {
    int layerSize = variables.layerSize;
    int steps = variables.steps;
    int N = variables.N;
    double tau = variables.tau;

//    double start = MPI_Wtime();
    // Разделяем параллелепипед по процессам на sub-параллелепипеды (Step 2)
    vector<ProcessParallelepiped> parallelepipeds = splitParallelepiped(N, variables.countOfProcesses);
//    double end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "Split time:  " << end - start << endl;
//    }

    // Запоминаем рабочий параллелепипед
    variables.processParallelepiped = parallelepipeds[variables.processId];

//    start = MPI_Wtime();
    // Заполняем соседей
    fillNeighbours(parallelepipeds, variables);
//    end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "fillNeighbours time:  " << end - start << endl;
//        int total = 0;
//        for (int i = 0; i < parallelepipeds.size(); i++) {
//            cout << "Parallelepiped " << i << endl <<
//                 "; xMin: " << parallelepipeds[i].xMin << "; xMax: " << parallelepipeds[i].xMax <<
//                 "; yMin: " << parallelepipeds[i].yMin << "; yMax: " << parallelepipeds[i].yMax <<
//                 "; zMin: " << parallelepipeds[i].zMin << "; zMax: " << parallelepipeds[i].zMax <<
//                 "; dx: " << parallelepipeds[i].dx << "; dy: " << parallelepipeds[i].dy << "; dz: "
//                 << parallelepipeds[i].dz <<
//                 "; size: " << parallelepipeds[i].size <<
//                 endl;
//            total += parallelepipeds[i].size;
//        }
//
//        cout << "N*N*N: " << variables.N * variables.N * variables.N << "; total size: " << total << endl;
//        cout << "parallelepipeds.size(): " << parallelepipeds.size() << endl;
//    }

    vector<vector<double>> u(3, vector<double>(variables.processParallelepiped.size));

//    start = MPI_Wtime();
    // Заполняем начальнные условия (step 4)
    fillVectorByInitialValues(u, variables);
//    end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "fillVectorByInitialValues time:  " << end - start << endl;
//    }

//    start = MPI_Wtime();
    double error0 = evaluateError(u[0], 0, variables);
//    end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "error0 time:  " << end - start << endl;
//    }
//    start = MPI_Wtime();
    double error1 = evaluateError(u[1], variables.tau, variables);
//    end = MPI_Wtime();
//    if (variables.processId == MAIN_PROCESS_ID) {
//        cout << "error1 time:  " << end - start << endl;
//    }

    if (variables.processId == MAIN_PROCESS_ID) {
        cout << "Layer 0 max error: " << error0 << endl;
        cout << "Layer 1 max error: " << error1 << endl;
    }

    double t = steps * tau;
    for (int step = 2; step <= steps; step++) {
//        start = MPI_Wtime();
        // Заполняем следующий слой u (step 5 + 6)
        fillNextLayer(u[(step + 1) % 3], u[(step + 2) % 3], u[step % 3], step * variables.tau, variables);
//        end = MPI_Wtime();
//        if (variables.processId == MAIN_PROCESS_ID) {
//            cout << "fillNextLayer time:  " << end - start << "; step: " << step << endl;
//        }

//        start = MPI_Wtime();
        // Вычисляем максимальную ошибку (step 8)
        double error = evaluateError(u[steps % 3], step * variables.tau, variables);
//        end = MPI_Wtime();
        if (variables.processId == MAIN_PROCESS_ID) {
//            cout << "Layer " << step << " max error: " << error << "; time: " << end - start << endl;
            cout << "Layer " << step << " max error: " << error << endl;
        }
    }

    // for report.pdf (нарисовать графики)
    // saveValues(u[steps % 3], t, parallelepipeds, "numerical.json", variables);

    // fillDifferenceValues(u[steps % 3], t, variables);
    // saveValues(u[steps % 3], t, parallelepipeds, "difference.json", variables);

    // fillAnalyticalValues(u[0], t, variables);
    // saveValues(u[0], t, parallelepipeds, "analytical.json", variables);
    //

    return evaluateError(u[steps % 3], t, variables);
}

// ЛАЗАРЕВ В.А. / 628 группа / 2 вариант
int main(int argc, char *argv[]) {
    SolverVariables variables;

    int processId, countOfProcesses;
    // Инициализация MPI, создание группы процессов, создание области связи MPI_COMM_WORLD
    MPI_Init(NULL, NULL);
    // Определяем номер процесса (сохранится в переменную processId)
    MPI_Comm_rank(MPI_COMM_WORLD, &processId);
    // Определение числа процессов в области связи MPI_COMM_WORLD (сохранится в переменную countOfProcesses)
    MPI_Comm_size(MPI_COMM_WORLD, &countOfProcesses);

    initVariables(argc, argv, variables, processId, countOfProcesses);

    // Устанавливаем количество omp-потоков
    omp_set_num_threads(variables.ompThreadsCount);

    //omp_set_dynamic(0);

    double start = MPI_Wtime();
    double error = 0;
    int loops = 1;

    for (size_t i = 0; i < loops; i++) {
        error += makeSolution(variables);
    }

    double end = MPI_Wtime();
    double diffTime = end - start;

    double minTime, maxTime, avgTime;

    // Отбираем минимальный diffTime по всем процессам
    MPI_Reduce(&diffTime, &minTime, 1, MPI_DOUBLE, MPI_MIN, MAIN_PROCESS_ID, MPI_COMM_WORLD);
    // Отбираем максимальный diffTime по всем процессам
    MPI_Reduce(&diffTime, &maxTime, 1, MPI_DOUBLE, MPI_MAX, MAIN_PROCESS_ID, MPI_COMM_WORLD);
    // Отбираем средний diffTime по всем процессам
    MPI_Reduce(&diffTime, &avgTime, 1, MPI_DOUBLE, MPI_SUM, MAIN_PROCESS_ID, MPI_COMM_WORLD);

    avgTime = avgTime / countOfProcesses;

    // Чтоб печатал информацию только один процесс
    if (processId == MAIN_PROCESS_ID) {
        ofstream fout("results.txt");
        fout << "### Lx = Ly = Lz = " << variables.L.x << ", N = " << variables.N << ", K = " << variables.K << endl
             << endl;
        fout << "| Число MPI процессов (P) | Время решения (с) | Ускорение | Погрешность |" << endl;
        fout << "|                     :-: |               :-: |       :-: |         :-: |" << endl;

        fout << "| " << setw(23) << variables.countOfProcesses;
        fout << " | " << setw(17) << (maxTime / loops);
        fout << " | " << "         ";
        fout << " | " << setw(11) << (error / loops);
        fout << " |" << endl;

        fout << "OMP threads: " << variables.ompThreadsCount << endl;
        fout << "Final error: " << error / loops << endl;
        fout << "Minimal time (s): " << minTime / loops << endl;
        fout << "Maximum time (s): " << maxTime / loops << endl;
        fout << "Average time (s): " << avgTime / loops << endl << endl;
        fout.close();
    }

    MPI_Finalize();

    return 0;
}